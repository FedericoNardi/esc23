
#include <hip/hip_runtime.h>
#include <iostream>
// Add an empty kernel
__global__ void mykernel()
{
}

int main()
{
    hipStream_t stream;
    hipStreamCreate(&stream);
    // launch the kernel on the stream
    mykernel<<<1, 1, 0, stream>>>(); // kernel launch is asynchronous << _#_blocks_, _#_threads_per_block, ...>>
    std::cout << "Hello, world!\n";
    hipStreamSynchronize(stream);
    hipStreamDestroy(stream);
}